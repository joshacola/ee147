#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void gpu_filter(ebmpBYTE* pixels, ebmpBYTE* pixels_out, int* weights, int width, int height){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    while (i<(width*height)){
        int column = i % width;
        int row = i / width;
        if (row > 0 && column > 0 && row < height - 1 && column < width - 1){
            int Red = 0;
            int Green = 0;
            int Blue = 0;
            for (int j = column-1; j < column + 2; j++){
                Blue += pixels[((row-1)*width+j)*3+0]*weights[j+1-column]/weights[j+10-column];
                Green += pixels[((row-1)*width+j)*3+1]*weights[j+1-column]/weights[j+10-column];
                Red += pixels[((row-1)*width+j)*3+2]*weights[j+1-column]/weights[j+10-column];

                Blue += pixels[(row*width+j)*3+0]*weights[j+4-column]/weights[j+13-column];
                Green += pixels[(row*width+j)*3+1]*weights[j+4-column]/weights[j+13-column];
                Red += pixels[(row*width+j)*3+2]*weights[j+4-column]/weights[j+13-column];

                Blue += pixels[((row+1)*width+j)*3+0]*weights[j+7-column]/weights[j+16-column];
                Green += pixels[((row+1)*width+j)*3+1]*weights[j+7-column]/weights[j+16-column];
                Red += pixels[((row+1)*width+j)*3+2]*weights[j+7-column]/weights[j+16-column];
            }
	    if (Blue > 255) Blue = 255;
            if (Green > 255) Green = 255;
            if (Red > 255) Red = 255;
            if (Blue < 0) Blue = 0;
            if (Green < 0) Green = 0;;
            if (Red < 0) Red = 0;
	    pixels_out[(row*width+column)*3] = Blue;
	    pixels_out[(row*width+column)*3+1] = Green;
	    pixels_out[(row*width+column)*3+2] = Red;
        }
        i+=1024;
    }
    return;
}
