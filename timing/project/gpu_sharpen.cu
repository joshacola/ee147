#include <string>

void gpu_sharpen (std::string filename)
{
    BMP Background;
    Background.ReadFromFile(filename.c_str());
    int height = Background.TellHeight();
    int width = Background.TellWidth();
    int depth = Background.TellBitDepth();
    BMP Output(Background);

    hipError_t cuda_ret;

    int *weights_d;

    int weights[18];


//SET WEIGHTS:
    for(int i = 9; i < 18; i++){
        weights[i] = 1;
    }
    weights[0] = 0;
    weights[1] = -1;
    weights[2] = 0;
    weights[3] = -1;
    weights[4] = 5;
    weights[5] = -1;
    weights[6] = 0;
    weights[7] = -1;
    weights[8] = 0;

//WEIGHTS SET

    ebmpBYTE *A_h, *B_h;
    ebmpBYTE *A_d, *B_d;
    A_h = (ebmpBYTE*) malloc( sizeof(ebmpBYTE)*width*height*3 );
    B_h = (ebmpBYTE*) malloc( sizeof(ebmpBYTE)*width*height*3 );
    for (int i = 0; i < height; i++){
	for (int j = 0; j < width; j++){
	    A_h[(i*width+j)*3] = Background.Pixels[i][j].Blue;
	    A_h[(i*width+j)*3+1] = Background.Pixels[i][j].Green;
	    A_h[(i*width+j)*3+2] = Background.Pixels[i][j].Red;

	}
    }
    dim3 dim_grid, dim_block;

    hipMalloc((void**)&weights_d, sizeof(int)*18 );
    hipMalloc((void**)&A_d, sizeof(ebmpBYTE)*width*height*3);
    hipMalloc((void**)&B_d, sizeof(ebmpBYTE)*width*height*3);

    hipDeviceSynchronize();

    hipMemcpy(weights_d, &weights[0], sizeof(int)*18, hipMemcpyHostToDevice);
    hipMemcpy(A_d, A_h, sizeof(ebmpBYTE)*width*height*3, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

//Timing start
    hipEvent_t begin, end;
    float time;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    hipEventRecord(begin, 0);

    dim3 DimGrid(1, 1, 1);
    dim3 DimBlock(1024, 1, 1);

    gpu_filter<<<DimGrid, DimBlock>>>(A_d, B_d, weights_d, width, height);

    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) printf("error");

    hipMemcpy(B_h, B_d, sizeof(ebmpBYTE)*width*height*3, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    for (int i = 0; i < height; i++){
	for (int j = 0; j < width; j++){
	    Output.Pixels[i][j].Blue = B_h[(i*width+j)*3];
	    Output.Pixels[i][j].Green = B_h[(i*width+j)*3+1];
	    Output.Pixels[i][j].Red = B_h[(i*width+j)*3+2];
	}
    }

//Timing end
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time, begin, end);
    printf("GPU Sharpen time: %f ms \n", time );

    std::string fileout = filename;
    fileout.pop_back();
    fileout.pop_back();
    fileout.pop_back();
    fileout.pop_back();
    string extra = "_gpu_sharpen.bmp";
    fileout = fileout + extra;
    Output.WriteToFile(fileout.c_str());
    free(A_h);
    free(B_h);

    hipFree(weights_d);
    hipFree(B_d);
    hipFree(A_d);
    return;
}


